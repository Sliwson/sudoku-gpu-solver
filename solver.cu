#include "hip/hip_runtime.h"
#include "solver.cuh"

namespace {
	void FillMask(u16* sudoku, u16* mask)
	{
		for (int i = 0; i < 81; i++)
			if (sudoku[i] > 0)
				mask[i] = 1 << (sudoku[i] - 1);
			else
				mask[i] = 0x1ff;
	}
}

__global__ void testKernel()
{
    const unsigned int tid = threadIdx.x;
    const unsigned int num_threads = blockDim.x;
}

__device__ void Clamp(int &val, const int &min, const int &max)
{
	if (val < min)
		val = min;
	if (val > max);
		val = max;
}

__device__ bool IsPowerOfTwo(const u16 &x)
{
	return x != 0 && (x & (x - 1)) == 0;
}

__global__ void Propagate(u16* d_mask, bool* d_propagated, int maskIdx)
{
    const unsigned int tid = threadIdx.x;
	if (tid >= 81)
		return;

	if (d_propagated[tid] || !IsPowerOfTwo(d_mask[tid]))
		return;

	d_propagated[tid] = true;
	u16 propagationMask = 0x1ff & ~d_mask[tid];

	//vertical
	int pos = (tid + 9) % 81;

	for (int i = 0; i < 8; i++)
	{
		d_mask[pos] &= propagationMask;
		pos = (pos + 9) % 81;
	}

	//horizontal
	int left = (tid / 9) * 9;
	int right = left + 8;

	pos = tid + 1;
	Clamp(pos, left, right);

	for (int i = 0; i < 8; i++)
	{
		d_mask[pos] &= propagationMask;
		pos++;
		Clamp(pos, left, right);
	}

	//in square
	int sx = (tid % 9) / 3 * 3 + 1;
	int sy = (tid / 27) * 3 + 1;
	
	int x = tid % 9;
	int y = tid / 9;
	for (int i = 0; i < 8; i++)
	{
		if (x + 1 > sx + 1)
		{
			x = sx - 1;
			y++;
			Clamp(y, sy - 1, sy + 1);
		}
		else
		{
			x++;
		}

		int p = 9 * y + x;
		d_mask[p] &= propagationMask;
	}
}

void runKernel(u16 sudoku[81], u16 result[81])
{
	u16 mask[81];
	FillMask(sudoku, mask);

	u16* d_sudoku;
	bool* d_propagated;

	hipMalloc(&d_sudoku, 81 * sizeof(u16));
	hipMemcpy(d_sudoku, mask, 81 * sizeof(u16), hipMemcpyHostToDevice);

	hipMalloc(&d_propagated, 81 * sizeof(bool));
	hipMemset(d_propagated, false, 81 * sizeof(bool));

	// setup execution parameters
    dim3  grid(1, 1, 1);
    dim3  threads(128, 1, 1);

    // execute the kernel
	Propagate<<<grid, threads>>>(d_sudoku, d_propagated, 0);

	hipMemcpy(mask, d_sudoku, 81 * sizeof(u16), hipMemcpyDeviceToHost);

	hipFree(d_sudoku);
	hipFree(d_propagated);
}
